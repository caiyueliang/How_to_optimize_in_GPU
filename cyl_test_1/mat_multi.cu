#include "hip/hip_runtime.h"
// 矩阵类型，行优先，M(row, col) = *(M.elements + row * M.width + col)
struct Matrix
{
    int width;
    int height;
    float *elements;
};

// 获取矩阵A的(row, col)元素
__device__ float getElement(Matrix *A, int row, int col)
{
    return A->elements[row * A->width + col];
}

// 为矩阵A的(row, col)元素赋值
__device__ void setElement(Matrix *A, int row, int col, float value)
{
    A->elements[row * A->width + col] = value;
}

// 矩阵相乘kernel，2-D，每个线程计算一个元素
__global__ void matMulKernel(Matrix *A, Matrix *B, Matrix *C)
{
    float Cvalue = 0.0;
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    for (int i = 0; i < A->width; ++i)
    {
        Cvalue += getElement(A, row, i) * getElement(B, i, col);
    }
    setElement(C, row, col, Cvalue);
}

int main(int argc, char **argv)
{
    // 检查是否有足够的命令行参数
    if (argc < 3) {
        std::cerr << "Usage: " << argv[0] << " <block_len> <epoch>" << std::endl;
        return 1;
    }

    // 获取并打印传入的参数
    int block_len = std::atoi(argv[1]);
    int epoch = std::atoi(argv[2]);
    std::cout << "Parameter [block_len]: " << block_len << std::endl;
    std::cout << "Parameter [epoch]: " << epoch << std::endl;

    int width = 1 << 10;
    int height = 1 << 10;
    Matrix *A, *B, *C;
    // 申请托管内存
    hipMallocManaged((void**)&A, sizeof(Matrix));      // Matrix 也要用 hipMallocManaged 申请内存。
    hipMallocManaged((void**)&B, sizeof(Matrix));
    hipMallocManaged((void**)&C, sizeof(Matrix));
    int nBytes = width * height * sizeof(float);
    hipMallocManaged((void**)&A->elements, nBytes);    // Matrix 中的 elements 也要用 hipMallocManaged 申请内存。
    hipMallocManaged((void**)&B->elements, nBytes);
    hipMallocManaged((void**)&C->elements, nBytes);

    // 初始化数据
    A->height = height;
    A->width = width;
    B->height = height;
    B->width = width;
    C->height = height;
    C->width = width;
    for (int i = 0; i < width * height; ++i)
    {
        A->elements[i] = 1.0;
        B->elements[i] = 2.0;
    }

    // 定义kernel的执行配置
    // dim3 blockSize(32, 32);     // 32 * 32 = 1024
    dim3 blockSize(block_len, block_len);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, 
        (height + blockSize.y - 1) / blockSize.y);
    printf("Grid : {%d, %d, %d} blocks. Blocks : {%d, %d, %d} threads.\n",
        gridSize.x, gridSize.y, gridSize.z, blockSize.x, blockSize.y, blockSize.z);

    // 执行kernel
    matMulKernel << < gridSize, blockSize >> >(A, B, C);


    // 同步device 保证结果能正确访问
    hipDeviceSynchronize();
    // 检查执行结果
    float maxError = 0.0;
    for (int i = 0; i < width * height; ++i)
        maxError = fmax(maxError, fabs(C->elements[i] - 2 * width));
    std::cout << "最大误差: " << maxError << std::endl;

    return 0;
}