#include <bits/stdc++.h>
#include <hip/hip_runtime.h>


#include <time.h>
#include <sys/time.h>

#define THREAD_PER_BLOCK 256

__global__ void reduce0(float*vec_in, float*vec_out) {
    //__shared__ float* shared_vec = THREAD_PER_BLOCK * sizeof(float);
    __shared__ float shared_vec[THREAD_PER_BLOCK];          // 由__shared__修饰的变量。block内的线程共享。

    int id = threadIdx.x;
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    printf("threadIdx.x:%d = id:%d ; blockDim.x:%d * blockIdx.x:%d + threadIdx.x:%d = tid:%d\n", 
            threadIdx.x, id, blockDim.x, blockIdx.x, threadIdx.x, tid);
    // printf("threadIdx.x:%d + blockIdx.x:%d * blockDim.x:%d = index:%d\n", 
    //         threadIdx.x, blockIdx.x, blockDim.x, index);
    // printf("blockDim.x:%d * gridDim.x:%d = stride:%d\n",
    //         blockDim.x, gridDim.x, stride);
    shared_vec[id] = vec_in[tid];
    __syncthreads();

    for (int n = 1; n < blockDim.x; n = n * 2) {
        if (id % n == 0) {
            shared_vec[id] = shared_vec[id] + shared_vec[id + n];
        }
        __syncthreads();
    }

    if (tid % blockDim.x == 0) {
        printf("[last] id:%d ; tid:%d\n", id, tid);
        vec_out[id] = shared_vec[id];
    }
}

bool check(float *out, float *res, int n) {
    for (int i=0; i<n; i++) {
        if (out[i] != res[i])
            return false;
    }
    return true;
}

int main(){
    //const int N = 32 * 1024 * 1024;
    const int N = 2 * 256;
    int nBytes = N * sizeof(float);
    printf("N: %d, nBytes: %d \n", N, nBytes);

    // float *a = (float *)malloc(N*sizeof(float));
    // float *d_a;
    // cudaMalloc((void **)&d_a,N*sizeof(float));
    float *a;
    hipMallocManaged((void**)&a, nBytes);

    int block_num = N / THREAD_PER_BLOCK;   // 128 * 1024
    // float *out=(float *)malloc((N/THREAD_PER_BLOCK)*sizeof(float));
    // float *d_out;
    // cudaMalloc((void **)&d_out,(N/THREAD_PER_BLOCK)*sizeof(float));
    float *res=(float *)malloc(block_num * sizeof(float));
    float *out;
    hipMallocManaged((void**)&out, block_num * sizeof(float));

    for(int i=0; i<N; i++) {
        a[i]=1;
    }

    for(int i=0; i<block_num; i++) {
        float cur=0;
        for(int j=0; j<THREAD_PER_BLOCK; j++) {
            cur+=a[i*THREAD_PER_BLOCK+j];
        }
        res[i]=cur;
    }
    printf("res[0]: %f, res[1]: %f \n", res[0], res[1]);

    //cudaMemcpy(d_a,a,N*sizeof(float),cudaMemcpyHostToDevice);

    dim3 Grid(block_num, 1);            // {131072, 1, 1}
    dim3 Block(THREAD_PER_BLOCK, 1);    // {256, 1, 1}
    printf("Grid : {%d, %d, %d} blocks. Blocks : {%d, %d, %d} threads.\n",
        Grid.x, Grid.y, Grid.z, Block.x, Block.y, Block.z);

    reduce0<<<Grid, Block>>>(a, out);

    //cudaMemcpy(out,d_out,block_num*sizeof(float),cudaMemcpyDeviceToHost);
    hipDeviceSynchronize();

    if(check(out, res, block_num)) {
        printf("the ans is right\n");
    }
    else {
        printf("the ans is wrong\n");
        for(int i=0; i<block_num; i++) {
            printf("%lf ", out[i]);
        }
        printf("\n");
    }

    hipFree(a);
    hipFree(out);
}
